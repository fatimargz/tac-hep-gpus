#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>


#define BLOCK_SIZE 32

const int DSIZE = 256;
const int a = 1;
const int b = 1;

// error checking macro
#define cudaCheckErrors()                                       \
	do {                                                        \
		hipError_t __err = hipGetLastError();                 \
		if (__err != hipSuccess) {                             \
			fprintf(stderr, "Error:  %s at %s:%d \n",           \
			hipGetErrorString(__err),__FILE__, __LINE__);      \
			fprintf(stderr, "*** FAILED - ABORTING***\n");      \
			exit(1);                                            \
		}                                                       \
	} while (0)


// CUDA kernel that runs on the GPU
__global__ void dot_product(const int *A, const int *B, int *C, int N) {

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx < N){
                atomicAdd(C,A[i] * B[i]);
        }

}


int main() {
	
	// Create the device and host pointers
	int *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

	// Fill in the host pointers 
	h_A = new int[DSIZE];
	h_B = new int[DSIZE];
	h_C = new int;
	for (int i = 0; i < DSIZE; i++){
		h_A[i] = a;
		h_B[i] = b;
	}

	*h_C = 0;


	// Allocate device memory 
	hipMalloc(&d_A, sizeof(int)*DSIZE);
        hipMalloc(&d_B,sizeof(int)*DSIZE);
        hipMalloc($d_C,sizeof(int));

	// Check memory allocation for errors
	cudaCheckErrors();

	// Copy the matrices on GPU
	hipMemcpy(&d_A, sizeof(int)*DSIZE, hipMemcpyHostToDevice);
        hipMemcpy(&d_B, sizeof(int)*DSIZE, hipMemcpyHostToDevice);
        hipMemcpy(&d_C, sizeof(int), hipMemcpyHostToDevice);

	// Check memory copy for errors
	cudaCheckErrors();

	// Define block/grid dimentions and launch kernel
	dim3 blockSize(BLOCK_SIZE);  // 32 threads in x, 16 threads in y
        dim3 gridSize((DSIZE+BLOCK_SIZE-1)/BLOCK_SIZE);
	dot_product<<<grid_size,block_size>>>(d_A, d_B, d_C, DSIZE);

	// Copy results back to host
	 hipMemcpy(h_C,d_C, sizeof(int), hipMemcpyDeviceToHost);

    // Check copy for errors
	 cudaCheckErrors();

	// Verify result
	std::cout<<*h_A,*h_B,*h_C<<std::endl;

	// Free allocated memory
	free(h_A);
        free(h_B);
        free(h_C);

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);


	return 0;

}
