
#include <hip/hip_runtime.h>
#include <stdio.h>


const int DSIZE = 40960;
const int block_size = 256;
const int grid_size = DSIZE/block_size;


__global__ void swap_vectors(float *A, float *B, int v_size) {

    //FIXME:
    // Express the vector index in terms of threads and blocks
    int idx =  threadIdx.x + blockDim.x * blockIdx.x;
    // Swap the vector elements - make sure you are not out of range
    if (idx < v_size){
	    float a = A[idx];
	    A[idx] = B[idx];
	    B[idx] = a;
    }
}


int main() {


    float *h_A, *h_B, *d_A, *d_B;
    h_A = new float[DSIZE];
    h_B = new float[DSIZE];


    for (int i = 0; i < DSIZE; i++) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }
    // Print old vectors
    printf("before: A[0] = %f\n", h_A[0]);
    printf("before: B[0] = %f\n", h_B[0]);



    // Allocate memory for host and device pointers 
    hipMalloc(&d_A, DSIZE*sizeof(float));
    hipMalloc(&d_B, DSIZE*sizeof(float));
    
    // Copy from host to device
    hipMemcpy(d_A, h_A, DSIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE*sizeof(float), hipMemcpyHostToDevice);
    // Launch the kernel
    swap_vectors<<<grid_size, block_size>>>(d_A, d_B, DSIZE);

    // Copy back to host 
    hipMemcpy(h_A, d_A, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, DSIZE*sizeof(float), hipMemcpyDeviceToHost);

    // Print and check some elements to make sure swapping was successfull
    printf("after: A[0] = %f\n", h_A[0]);
    printf("after: B[0] = %f\n", h_B[0]);


    // Free the memory 
    free(h_A);
    free(h_B);
 
    hipFree(d_A);
    hipFree(d_B);
    
}
