
#include <hip/hip_runtime.h>
#include <stdio.h>


const int DSIZE_X = 256;
const int DSIZE_Y = 256;

__global__ void add_matrix(const float *A, const float *B, float *C, int Nx, int Ny)
{
    //FIXME:
    // Express in terms of threads and blocks
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    // Add the two matrices - make sure you are not out of range
    if (idx <  Nx && idy < Ny ){
	int index = idy*Nx + idx;
        C[index] =  A[index] + B[index];
    }
}

int main()
{

    // Create and allocate memory for host and device pointers 
    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
    h_A = new float[DSIZE_X * DSIZE_Y];
    h_B = new float[DSIZE_X * DSIZE_Y];
    h_C = new float[DSIZE_X * DSIZE_Y];

    // Fill in the matrices
    for (int i = 0; i < DSIZE_X; i++) {
        for (int j = 0; j < DSIZE_Y; j++) {
            int ind = i*DSIZE_Y + j;
            h_A[ind] = rand()/(float)RAND_MAX;
	    h_B[ind] = rand()/(float)RAND_MAX;
        }
    }

    // Copy from host to device
    hipMalloc(&d_A, DSIZE_X * DSIZE_Y * sizeof(float));
    hipMalloc(&d_B, DSIZE_X * DSIZE_Y * sizeof(float));
    hipMalloc(&d_C, DSIZE_X * DSIZE_Y * sizeof(float));
    hipMemcpy(d_A, h_A, DSIZE_X * DSIZE_Y * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE_X * DSIZE_Y * sizeof(float), hipMemcpyHostToDevice);
    // Launch the kernel
    // dim3 is a built in CUDA type that allows you to define the block 
    // size and grid size in more than 1 dimentions
    // Syntax : dim3(Nx,Ny,Nz)
    dim3 blockSize(16,16); 
    dim3 gridSize((DSIZE_X + blockSize.x-1)/blockSize.x,(DSIZE_Y + blockSize.y -1)/blockSize.y); 
    
    add_matrix<<<gridSize, blockSize>>>(d_A, d_B, d_C, DSIZE_X, DSIZE_Y);
    
    // Copy back to host 
    hipMemcpy(h_C, d_C, DSIZE_X * DSIZE_Y * sizeof(float), hipMemcpyDeviceToHost);

    // Print and check some elements to make the addition was succesfull
    printf("A[0,0]] = %f\n", h_A[0]);
    printf("B[0,0] = %f\n", h_B[0]);
    printf("C[0,0] = %f\n", h_C[0]);
    // Free the memory     

    free(h_A);
    free(h_B);
    free(h_C);


    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}
