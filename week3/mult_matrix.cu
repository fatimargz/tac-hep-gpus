
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

const int DSIZE = 256;
const float A_val = 3.0f;
const float B_val = 2.0f;

// error checking macro
#define cudaCheckErrors(msg)                                   \
   do {                                                        \
       hipError_t __err = hipGetLastError();                 \
       if (__err != hipSuccess) {                             \
           fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n",  \
                   msg, hipGetErrorString(__err),             \
                   __FILE__, __LINE__);                        \
           fprintf(stderr, "*** FAILED - ABORTING\n");         \
           exit(1);                                            \
       }                                                       \
   } while (0)

// Square matrix multiplication on CPU : C = A * B
void matrix_mul_cpu(const float *A, const float *B, float *C, int size) {
  //FIXME:
  for (int i=0; i<size; i++){
	 for (int j=0; j<size; j++){
		 float temp = 0;
		 for (int k = 0; k < size; k++){
			 temp += A[i*size + k] * B[k*size+j];
		 }
		 C[i*size+j] = temp;
	  }
  }
}

// Square matrix multiplication on GPU : C = A * B
__global__ void matrix_mul_gpu(const float *A, const float *B, float *C, int size) {

    //FIXME:
    // create thread x index
    // create thread y index
    int idx = threadIdx.x + blockDim.x * blockIdx.x; 
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    // Make sure we are not out of range
    if ((idx < size) && (idy < size)) {
        float temp = 0;
        for (int i = 0; i < size; i++){
            //FIXME : Add dot product of row and column
	    temp += A[idy*size+idx] * B [idy*size+idx];
        }
        C[idy*size+idx] = temp;                    
    }

}

int main() {

    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

    // These are used for timing
    clock_t t0, t1, t2, t3;
    double t1sum=0.0;
    double t2sum=0.0;
    double t3sum=0.0;

    // start timing
    t0 = clock();

    // N*N matrices defined in 1 dimention
    // If you prefer to do this in 2-dimentions cupdate accordingly
    h_A = new float[DSIZE*DSIZE];
    h_B = new float[DSIZE*DSIZE];
    h_C = new float[DSIZE*DSIZE];
    for (int i = 0; i < DSIZE*DSIZE; i++){
        h_A[i] = A_val;
        h_B[i] = B_val;
        h_C[i] = 0;
    }

    // Initialization timing
    t1 = clock();
    t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
    printf("Init took %f seconds.  Begin compute\n", t1sum);

    // Allocate device memory and copy input data from host to device
    hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));
    //FIXME:Add all other allocations and copies from host to device
    hipMalloc(&d_B, DSIZE*DSIZE*sizeof(float));
    hipMalloc(&d_C, DSIZE*DSIZE*sizeof(float));
    hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);


    // Launch kernel
    // Specify the block and grid dimentions 
    dim3 blockSize(16,16);  //FIXME
    dim3 grid((DSIZE + blockSize.x -1)/blockSize.x,(DSIZE+blockSize.y-1)/blockSize.y); //FIXME
    matrix_mul_gpu<<<grid, blockSize>>>(d_A, d_B, d_C, DSIZE);

    // Copy results back to host
    hipMemcpy(h_C, d_C, DSIZE*DSIZE*sizeof(float), hipMemcpyDeviceToHost);

    // GPU timing
    t2 = clock();
    t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
    printf ("Done. Compute took %f seconds\n", t2sum);
    
    // FIXME
    // Excecute and time the cpu matrix multiplication function
    matrix_mul_cpu(h_A, h_B, h_C, DSIZE);

    // CPU timing
    t3 = clock();
    t3sum = ((double)(t3-t2))/CLOCKS_PER_SEC;
    printf ("Done. Compute took %f seconds\n", t3sum);
	    
    // FIXME
    
    // Free memory 
    free(h_A);
    free(h_B);
    free(h_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;

}
